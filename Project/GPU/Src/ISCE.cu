#include "hip/hip_runtime.h"
//! Special relativistic dissipative hydro model - ISCE
/*!
    This script contains the function definitions for the ISCE model. The form
  of the quations has been taken from Dionysopoulou and we use a divergence cleaning method
  taken from Muddle.
    For detailed documentation about the methods contained herein, see ISCE.h
  and model.h.
*/

#include <stdexcept>
#include <cstdio>
#include "ISCE.h"
//#include "cminpack.h"
#include "cudaErrorCheck.h"

#define TOL 1.0e-12
#define EPS 1.0e-4
#define MAXITER 50

// Syntax
Data * d(this->data);

// Macro for getting array index
#define ID(variable, idx, jdx, kdx) ((variable)*(d->Nx)*(d->Ny)*(d->Nz) + (idx)*(d->Ny)*(d->Nz) + (jdx)*(d->Nz) + (kdx))
#define IDCons(var, idx, jdx, kdx) ( (var) + (idx)*(d->Ncons)*(d->Nz)*(d->Ny) + (jdx)*(d->Ncons)*(d->Nz) + (kdx)*(d->Ncons)  )
#define IDPrims(var, idx, jdx, kdx) ( (var) + (idx)*(d->Nprims)*(d->Nz)*(d->Ny) + (jdx)*(d->Nprims)*(d->Nz) + (kdx)*(d->Nprims)  )
#define IDAux(var, idx, jdx, kdx) ( (var) + (idx)*(d->Naux)*(d->Nz)*(d->Ny) + (jdx)*(d->Naux)*(d->Nz) + (kdx)*(d->Naux)  )

// C2P residual and rootfinder (Serial)
static double residual(const double, const double, const double, const double, double);
static int newton(double *Z, const double S_sqrd, const double D, const double Tau, double gamma, int i, int j, int k);

// C2P residual and rootfinder (Parallel)
__device__
static double residualParallel(const double Z, const double StildeSq, const double D, const double Tau, double gamma);
__device__
static int newtonParallel(double *Z, const double S_sqrd, const double D, const double Tau, double gamma);
__global__
static void getPrimitiveVarsParallel(double *cons, double *prims, double *aux, double *guess, int stream, double gamma, double sigma, int Ncons, int Nprims, int Naux, int origWidth, int streamWidth);

// enums to save looking up numbering of C/P/As when using ID accessor.
enum Cons { D, S1, S2, S3, Tau };
enum Prims { v1, v2, v3, p, rho, n, q1, q2, q3, Pi, pi11, pi12, pi13, pi22, pi23, pi33 };
enum Aux { h, T, e, W, q0, qv, pi00, pi01, pi02, pi03, Theta, vsqrd, rhohWsq, S_sqrd,
            q1NS, q2NS, q3NS, PiNS, pi11NS, pi12NS, pi13NS, pi22NS, pi23NS, pi33NS,
            q1LO, q2LO, q3LO, PiLO, pi11LO, pi12LO, pi13LO, pi22LO, pi23LO, pi33LO,  
            a1, a2, a3 };
enum TDerivs { dtp = 37, dtrho, dtn, dtv1, dtv2, dtv3, dtW, dtT, dtq1NS, dtq2NS, dtq3NS, dtPiNS,
            dtpi11NS, dtpi12NS, dtpi13NS, dtpi22NS, dtpi23NS, dtpi33NS, dtD, dtS1, dtS2, dtS3,
            dtTau, dtE};

ISCE::ISCE() : Model()
{
  modType_t = ModelType::ISCE;
  this->Ncons = 5;
  this->Nprims = 16;
  this->Naux = 59;

  hipHostAlloc((void **)&singleCons, sizeof(double) * this->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singlePrims, sizeof(double) * this->Nprims,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singleAux, sizeof(double) * this->Naux,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singleSource, sizeof(double) * this->Ncons,
                hipHostMallocPortable);
}

ISCE::~ISCE()
{
  // Free up
  hipHostFree(singleCons);
  hipHostFree(singlePrims);
  hipHostFree(singleAux);
  hipHostFree(singleSource);

  delete c2pArgs;
}

ISCE::ISCE(Data * data) : Model(data)
{
  modType_t = ModelType::ISCE;
  this->Ncons = (this->data)->Ncons = 5;
  this->Nprims = (this->data)->Nprims = 16;
  this->Naux = (this->data)->Naux = 61;

  // 0
  this->data->primsLabels.push_back("v1");   this->data->primsLabels.push_back("v2");
  this->data->primsLabels.push_back("v3");
  // 3
  this->data->primsLabels.push_back("p");   this->data->primsLabels.push_back("rho");
  this->data->primsLabels.push_back("n");   
  // 6
  this->data->primsLabels.push_back("q1");  this->data->primsLabels.push_back("q2");
  this->data->primsLabels.push_back("q3");
  // 9
  this->data->primsLabels.push_back("Pi");  
  // 10
  this->data->primsLabels.push_back("pi11");   this->data->primsLabels.push_back("pi12");
  this->data->primsLabels.push_back("pi13");  this->data->primsLabels.push_back("pi22");
  this->data->primsLabels.push_back("pi23");  this->data->primsLabels.push_back("pi33");

  // 0
  this->data->auxLabels.push_back("h");     this->data->auxLabels.push_back("T");
  this->data->auxLabels.push_back("e");     this->data->auxLabels.push_back("W");
  // 4
  this->data->auxLabels.push_back("q0");    this->data->auxLabels.push_back("qv");
  this->data->auxLabels.push_back("pi00");  this->data->auxLabels.push_back("pi01");
  this->data->auxLabels.push_back("pi02");  this->data->auxLabels.push_back("pi03");
  this->data->auxLabels.push_back("Theta"); 
  // 11
  this->data->auxLabels.push_back("vsqrd"); this->data->auxLabels.push_back("rhohWsq");
  this->data->auxLabels.push_back("S_sqrd");
  // 12
  this->data->auxLabels.push_back("q1NS");  this->data->auxLabels.push_back("q2NS");
  this->data->auxLabels.push_back("q3NS");
  // 15
  this->data->auxLabels.push_back("PiNS");    
  // 16
  this->data->auxLabels.push_back("pi11NS"); this->data->auxLabels.push_back("pi12NS");
  this->data->auxLabels.push_back("pi13NS"); this->data->auxLabels.push_back("pi22NS");
  this->data->auxLabels.push_back("pi23NS"); this->data->auxLabels.push_back("pi33NS");
  // 22
  this->data->auxLabels.push_back("q1LO");  this->data->auxLabels.push_back("q2LO");
  this->data->auxLabels.push_back("qLO");
  // 25
  this->data->auxLabels.push_back("PiLO");    
  // 26
  this->data->auxLabels.push_back("pi11LO"); this->data->auxLabels.push_back("pi12LO");
  this->data->auxLabels.push_back("pi13LO"); this->data->auxLabels.push_back("pi22LO");
  this->data->auxLabels.push_back("pi23LO"); this->data->auxLabels.push_back("pi33LO");
  // 32
  this->data->auxLabels.push_back("a1");     this->data->auxLabels.push_back("a2");   
  this->data->auxLabels.push_back("a3");

  // 35
  this->data->auxLabels.push_back("dtp");  this->data->auxLabels.push_back("dtrho");
  this->data->auxLabels.push_back("dtn");
  // 38
  this->data->auxLabels.push_back("dtv1");
  this->data->auxLabels.push_back("dtv2");  this->data->auxLabels.push_back("dtv3");
  // 41
  this->data->auxLabels.push_back("dtW");   this->data->auxLabels.push_back("dtT"); 
  // 43
  this->data->auxLabels.push_back("dtq1NS");  this->data->auxLabels.push_back("dtq2NS");
  this->data->auxLabels.push_back("dtq3NS");
  // 46
  this->data->auxLabels.push_back("dtPiNS");    
  // 47
  this->data->auxLabels.push_back("dtpi11NS"); this->data->auxLabels.push_back("dtpi12NS");
  this->data->auxLabels.push_back("dtpi13NS"); this->data->auxLabels.push_back("dtpi22NS");
  this->data->auxLabels.push_back("dtpi23NS"); this->data->auxLabels.push_back("dtpi33NS");
  // 53
  this->data->auxLabels.push_back("dtD"); this->data->auxLabels.push_back("dtS1");
  this->data->auxLabels.push_back("dtS2"); this->data->auxLabels.push_back("dtS3");
  this->data->auxLabels.push_back("dtTau");  this->data->auxLabels.push_back("dtE"); 

  // Single cell work arrays
  hipHostAlloc((void **)&singleCons, sizeof(double) * this->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singlePrims, sizeof(double) * this->Nprims,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singleAux, sizeof(double) * this->Naux,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singleSource, sizeof(double) * this->Ncons,
                hipHostMallocPortable);

  c2pArgs = new C2PArgs(this->data);
}

void ISCE::fluxVector(double *cons, double *prims, double *aux, double *f, const int dir)
{
  // Syntax
  Data * d(this->data);

  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        // Dv
        f[ID(0, i, j, k)] = cons[ID(Cons::D, i, j, k)]*prims[ID(dir, i, j, k)];
        // Sv + ..
        for (int nvar(0); nvar < 3; nvar++) {
          f[ID(1+nvar, i, j, k)] = cons[ID(Cons::S1+nvar, i, j, k)]*prims[ID(dir, i, j, k)]; // + ( prims[ID(Prims::q1+dir, i, j, k)] * prims[ID(Prims::v1+nvar, i, j, k)]  
            // - aux[ID(Aux::qv, i, j, k)]*prims[ID(Prims::v1+nvar, i, j, k)]*prims[ID(Prims::v1+dir, i, j, k)] ) * aux[ID(Aux::W, i, j, k)];
          // (p+Pi)delta_ij
          if (dir == nvar) {
            f[ID(1+nvar, i, j, k)] += (prims[ID(Prims::p, i, j, k)]); // + prims[ID(Prims::Pi, i, j, k)]);
          }
        }
        /*
        //  pi^i_j  
        if (dir == 0) {
          for (int nvar(0); nvar < 3; nvar++) {
            f[ID(1+nvar, i, j, k)] += prims[ID(Prims::pi11+nvar, i, j, k)];
          }
        } else if (dir == 1) {
          f[ID(1, i, j, k)] += prims[ID(Prims::pi12, i, j, k)];
          f[ID(2, i, j, k)] += prims[ID(Prims::pi22, i, j, k)];
          f[ID(3, i, j, k)] += prims[ID(Prims::pi23, i, j, k)];
        } else if (dir == 2) {
          f[ID(1, i, j, k)] += prims[ID(Prims::pi13, i, j, k)];
          f[ID(2, i, j, k)] += prims[ID(Prims::pi23, i, j, k)];
          f[ID(3, i, j, k)] += prims[ID(Prims::pi33, i, j, k)];
        } else {
          throw std::runtime_error("Flux direction is not 0, 1 or 2");
        }
        */

        // (Tau+p)*v + ...
        f[ID(4, i, j, k)] = (cons[ID(Cons::Tau, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * prims[ID(dir, i, j, k)];
          // + (prims[ID(Prims::q1+dir, i, j, k)] - aux[ID(Aux::qv, i, j, k)]*prims[ID(Prims::v1+dir, i, j, k)])*aux[ID(Aux::W, i, j, k)]
          // + aux[ID(Aux::pi01+dir, i, j, k)];
      } // End k loop
    } // End j loop
  } // End i loop
}

void ISCE::sourceTermSingleCell(double *cons, double *prims, double *aux, double *source, int i, int j, int k)
{
  // Syntax
  // Data * d(this->data);

  // D
  source[0] = 0.0;
  // S1,2,3
  source[1] = 0.0; 
  source[2] = 0.0;
  source[3] = 0.0; 
  // Tau
  source[4] = 0.0;

}

void ISCE::sourceTerm(double *cons, double *prims, double *aux, double *source)
{
  for (int i(0); i < this->data->Nx; i++) {
    for (int j(0); j < this->data->Ny; j++) {
      for (int k(0); k < this->data->Nz; k++) {

        // D
        source[ID(Cons::D, i, j, k)] = 0.0;
        // source[ID(Cons::D, i, j, k)] = 0.0;
        // S1,2,3
        source[ID(Cons::S1, i, j, k)] = 0.0;
        source[ID(Cons::S2, i, j, k)] = 0.0;
        source[ID(Cons::S3, i, j, k)] = 0.0;
        // Tau
        source[ID(Cons::Tau, i, j, k)] = 0.0;        
      }
    }
  }
}

void ISCE::getPrimitiveVarsSingleCell(double *cons, double *prims, double *aux, int i, int j, int k)
{
    Data * d(this->data);

    // Sbarsq, tauBar
    aux[Aux::S_sqrd] = cons[Cons::S1] * cons[Cons::S1] + cons[Cons::S2] * cons[Cons::S2] + cons[Cons::S3] * cons[Cons::S3];
 
    // Solve
    newtonParallel(&prims[Prims::p], aux[Aux::S_sqrd], cons[Cons::D], cons[Cons::Tau], d->gamma);

    double E = cons[Cons::Tau] + cons[Cons::D];
    
    // vsq
    aux[Aux::vsqrd] = aux[Aux::S_sqrd] / ((E + prims[Prims::p]) * (E + prims[Prims::p]));
    // W
    aux[Aux::W] = 1.0 / sqrt(1 - aux[Aux::vsqrd]);
    // rho
    prims[Prims::n] = cons[Cons::D] / aux[Aux::W];
    // rho_plus_p
    double rho_plus_p = (E + prims[Prims::p]) / (aux[Aux::W] * aux[Aux::W]);
    // p  
    prims[Prims::p] = (rho_plus_p - prims[Prims::n]) / ((d->gamma-1)/d->gamma);
    // rho
    prims[Prims::rho] = rho_plus_p - prims[Prims::p];    
    // vx, vy, vz
    prims[Prims::v1] = cons[Cons::S1] / (rho_plus_p*aux[Aux::W] * aux[Aux::W]);
    prims[Prims::v1] = cons[Cons::S2] / (rho_plus_p*aux[Aux::W] * aux[Aux::W]);
    prims[Prims::v1] = cons[Cons::S3] / (rho_plus_p*aux[Aux::W] * aux[Aux::W]);

    aux[Aux::e] = prims[Prims::p] / (prims[Prims::n]*(d->gamma-1));
    aux[Aux::T] = prims[Prims::p] / prims[Prims::n];
    aux[Aux::h] = 1 + aux[Aux::e] + prims[Prims::p] / prims[Prims::n];

}

void ISCE::getPrimitiveVars(double *cons, double *prims, double *aux)
{
  // Syntax
  Data * d(this->data);

  // First need to copy data to the device
  // A single cell requires all cons variables and aux10 to start the guessing
  // Rearrange data into host arrays ready for copying
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        for (int var(0); var < d->Ncons; var++) {
          c2pArgs->cons_h[IDCons(var, i, j, k)] = cons[ID(var, i, j, k)];
        }
        c2pArgs->guess_h[ID(0, i, j, k)] = prims[ID(Prims::p, i, j, k)];
      }
    }
  }

  // Data is in correct order, now stream data to the device
  for (int i(0); i < c2pArgs->Nstreams; i++) {
    // Which cell is at the left bound?
    int lcell(i * c2pArgs->streamWidth);
    // Which cell is at the right bound?
    int rcell(lcell + c2pArgs->streamWidth);
    if (rcell > d->Ncells) rcell = d->Ncells;
    // Memory size to copy in
    int width(rcell - lcell);
    int inMemsize(width * sizeof(double));

    // Send stream's data
    gpuErrchk( hipMemcpyAsync(c2pArgs->cons_d[i], c2pArgs->cons_h + lcell*d->Ncons, inMemsize*d->Ncons, hipMemcpyHostToDevice, c2pArgs->stream[i]) );
    gpuErrchk( hipMemcpyAsync(c2pArgs->guess_d[i], c2pArgs->guess_h + lcell, inMemsize, hipMemcpyHostToDevice, c2pArgs->stream[i]) );

    // Call kernel and operate on data
    getPrimitiveVarsParallel <<< c2pArgs->bpg, c2pArgs->tpb,
        c2pArgs->tpb * c2pArgs->cellMem, c2pArgs->stream[i] >>> (c2pArgs->cons_d[i],
        c2pArgs->prims_d[i], c2pArgs->aux_d[i], c2pArgs->guess_d[i], i, d->gamma, d->sigma, d->Ncons,
        d->Nprims, d->Naux, c2pArgs->streamWidth, width);


    // Copy all data back
    gpuErrchk( hipMemcpyAsync(c2pArgs->prims_h + lcell*d->Nprims, c2pArgs->prims_d[i], inMemsize*d->Nprims, hipMemcpyDeviceToHost, c2pArgs->stream[i]) );
    gpuErrchk( hipMemcpyAsync(c2pArgs->aux_h + lcell*d->Naux, c2pArgs->aux_d[i], inMemsize*d->Naux, hipMemcpyDeviceToHost, c2pArgs->stream[i]) );
  }
  gpuErrchk( hipDeviceSynchronize() );

  // Rearrange data back into arrays
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {

        for (int var(0); var < d->Nprims; var++) {
          prims[ID(var, i, j, k)] = c2pArgs->prims_h[IDPrims(var, i, j, k)];
        }
        for (int var(0); var < d->Naux; var++) {
          aux[ID(var, i, j, k)] = c2pArgs->aux_h[IDAux(var, i, j, k)];
        }
      }
    }
  }
}

void ISCE::primsToAll(double *cons, double *prims, double *aux)
{
  // Syntax
  Data * d(this->data);

  // printf("Calling primsToAll\n");

  // W, q_kv^k, pi^0_0
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        aux[ID(Aux::vsqrd, i, j, k)] = prims[ID(Prims::v1, i, j, k)]*prims[ID(Prims::v1, i, j, k)] 
                                  + prims[ID(Prims::v2, i, j, k)]*prims[ID(Prims::v2, i, j, k)] 
                                  + prims[ID(Prims::v3, i, j, k)]*prims[ID(Prims::v3, i, j, k)];
        aux[ID(Aux::W, i, j, k)] = 1 / sqrt( 1 - aux[ID(Aux::vsqrd, i, j, k)] );
        aux[ID(Aux::e, i, j, k)] = prims[ID(Prims::p, i, j, k)] / (prims[ID(Prims::n, i, j, k)]*(d->gamma-1));
        prims[ID(Prims::rho, i, j, k)] = prims[ID(Prims::n, i, j, k)]*(1+aux[ID(Aux::e, i, j, k)]);
        aux[ID(Aux::T, i, j, k)] = prims[ID(Prims::p, i, j, k)] / prims[ID(Prims::n, i, j, k)];
        aux[ID(Aux::h, i, j, k)] = 1 + aux[ID(Aux::e, i, j, k)] + prims[ID(Prims::p, i, j, k)] / prims[ID(Prims::n, i, j, k)];
      }
    }
  }

  // Conserveds are now EulerSR form (no dissipation)
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        // D
        cons[ID(Cons::D, i, j, k)] = prims[ID(Prims::n, i, j, k)] * aux[ID(Aux::W, i, j, k)];
        // S1,2,3
        cons[ID(Cons::S1, i, j, k)] = (prims[ID(Prims::rho, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * aux[ID(Aux::W, i, j, k)]*aux[ID(Aux::W, i, j, k)] * prims[ID(Prims::v1, i, j, k)]; 
        cons[ID(Cons::S2, i, j, k)] = (prims[ID(Prims::rho, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * aux[ID(Aux::W, i, j, k)]*aux[ID(Aux::W, i, j, k)] * prims[ID(Prims::v2, i, j, k)]; 
        cons[ID(Cons::S3, i, j, k)] = (prims[ID(Prims::rho, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * aux[ID(Aux::W, i, j, k)]*aux[ID(Aux::W, i, j, k)] * prims[ID(Prims::v3, i, j, k)];
        // Tau
        cons[ID(Cons::Tau, i, j, k)] = (prims[ID(Prims::rho, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * aux[ID(Aux::W, i, j, k)]*aux[ID(Aux::W, i, j, k)] 
        - (prims[ID(Prims::p, i, j, k)] + prims[ID(Prims::n, i, j, k)] * aux[ID(Aux::W, i, j, k)]);
      }  
    }
  }
}

static double residual(const double Z, const double S_sqrd, const double D, const double tau, double gamma)
{
  // Decalre variables
  double v_sqrd, E, W, rho, p, n, rho_plus_p, resid;

  E = tau + D;
  v_sqrd = S_sqrd / ((E + Z)*(E + Z));

  // Sanity check
  if (v_sqrd >= 1.0 || Z < 0) return 1.0e6;

  W = 1/sqrt(1 - v_sqrd);
  n = D / W;
  rho_plus_p = (E + Z)/(W*W);
  p = (rho_plus_p - n)*((gamma-1)/gamma);
  rho = rho_plus_p - p;

  // Second sanity check
  if (rho < 0 || p < 0 || W < 1 ) return 1.0e6;

  // Values are physical, compute residual
  resid = p - Z;
  return resid;

}


static int newton(double *Z, const double S_sqrd, const double D, const double Tau, double gamma, int i, int j, int k)
{
  // Rootfind data
  double bestX;
  double x0(*Z);
  double eps(EPS);
  double x1(x0 + eps);
  double tol(TOL);
  double x2;
  double bestF;
  double f0(residual(x0, S_sqrd, D, Tau, gamma));
  double f1(residual(x1, S_sqrd, D, Tau, gamma));
  int iter;
  int maxiter(MAXITER);
  int found(0);

  // If root can not be found return the best so far
  bestX = x0; bestF = f0;
  for (iter=0; iter<maxiter; iter++) {
    if (fabs(f0) < tol) {
      *Z = x0;
      found = 1;
      break;
    }

    x2 = x1 - f1 * (x1 - x0) / (f1 - f0);
    x1 = x0;
    x0 = x2;
    f1 = f0;
    f0 = residual(x0, S_sqrd, D, Tau, gamma);
    if (fabs(f0) < fabs(bestF)) {
      bestX = x0;
      bestF = f0;
    }
  }

  if (!found) {
    // Store result of Z=p
    *Z = bestX;
    char s[200];
    sprintf(s, "C2P could not converge in cell (%d, %d, %d)\n", i, j, k);

    throw std::runtime_error(s);
  }
  return 1;
}

// /*!
//     This is the device version of the getPrimitiveVars that takes a streams data
//     and computes the rest of the prims and aux vars. This is called when
//     ISCE::getPrimitiveVars is required, i.e. all cells need to be found.
// */
__global__
static void getPrimitiveVarsParallel(double *streamCons, double *streamPrims, double *streamAux, double *guess, int stream, double gamma, double sigma, int Ncons, int Nprims, int Naux, int origWidth, int streamWidth)
{
  // First need thread indicies
  const int tID(threadIdx.x);                     //!< thread index (in block)
  const int lID(tID + blockIdx.x * blockDim.x);   //!< local index (in stream)
  // const int gID(lID + stream * origWidth);        //!< global index (in domain)
  // Allocate shared memory
  extern __shared__ double sharedArray [];
  double * cons = &sharedArray[tID * (Ncons + Nprims + Naux)];
  double * prims = &cons[Ncons];
  double * aux = &prims[Nprims];

  // enums to save looking up numbering of C/P/As when using ID accessor.
  // enum Cons { D, S1, S2, S3, Tau };
  // enum Prims { v1, v2, v3, p, rho, n, q1, q2, q3, Pi, pi11, pi12, pi13, pi22, pi23, pi33 };
  // enum Aux { h, T, e, W, q0, qv, pi00, pi01, pi02, pi03, Theta, vsqrd, rhohWsq, S_sqrd,
  //             q1NS, q2NS, q3NS, PiNS, pi11NS, pi12NS, pi13NS, pi22NS, pi23NS, pi33NS,
  //             q1LO, q2LO, q3LO, PiLO, pi11LO, pi12LO, pi13LO, pi22LO, pi23LO, pi33LO,  
  //             a1, a2, a3 };
  // enum TDerivs { dtp = 37, dtrho, dtn, dtv1, dtv2, dtv3, dtW, dtT, dtq1NS, dtq2NS, dtq3NS, dtPiNS,
  //             dtpi11NS, dtpi12NS, dtpi13NS, dtpi22NS, dtpi23NS, dtpi33NS, dtD, dtS1, dtS2, dtS3,
  //             dtTau, dtE};

  if (lID < streamWidth) {

    // Load conserved vector into shared memory, and the initial guess
    for (int i(0); i < Ncons; i++) cons[i] = streamCons[lID * Ncons + i];
    // p is guessed
    prims[3] = guess[lID];

    // Sbarsq, tauBar
    aux[Aux::S_sqrd] = cons[Cons::S1] * cons[Cons::S1] + cons[Cons::S2] * cons[Cons::S2] + cons[Cons::S3] * cons[Cons::S3];
 
    // Solve
    newtonParallel(&prims[Prims::p], aux[Aux::S_sqrd], cons[Cons::D], cons[Cons::Tau], gamma);

    double E = cons[Cons::Tau] + cons[Cons::D];
    
    // vsq
    aux[Aux::vsqrd] = aux[Aux::S_sqrd] / ((E + prims[Prims::p]) * (E + prims[Prims::p]));
    // W
    aux[Aux::W] = 1.0 / sqrt(1 - aux[Aux::vsqrd]);
    // rho
    prims[Prims::n] = cons[Cons::D] / aux[Aux::W];
    // rho_plus_p
    double rho_plus_p = (E + prims[Prims::p]) / (aux[Aux::W] * aux[Aux::W]);
    // p  
    prims[Prims::p] = (rho_plus_p - prims[Prims::n]) / ((gamma-1)/gamma);
    // rho
    prims[Prims::rho] = rho_plus_p - prims[Prims::p];    
    // vx, vy, vz
    prims[Prims::v1] = cons[Cons::S1] / (rho_plus_p*aux[Aux::W] * aux[Aux::W]);
    prims[Prims::v1] = cons[Cons::S2] / (rho_plus_p*aux[Aux::W] * aux[Aux::W]);
    prims[Prims::v1] = cons[Cons::S3] / (rho_plus_p*aux[Aux::W] * aux[Aux::W]);

    aux[Aux::e] = prims[Prims::p] / (prims[Prims::n]*(gamma-1));
    aux[Aux::T] = prims[Prims::p] / prims[Prims::n];
    aux[Aux::h] = 1 + aux[Aux::e] + prims[Prims::p] / prims[Prims::n];

  }

  // Copy data back from shared memory into device arrays
  for (int i(0); i < Nprims; i++) streamPrims[lID * Nprims + i] = prims[i];
  for (int i(0); i < Naux; i++) streamAux[lID * Naux + i] = aux[i];

}

__device__
static int newtonParallel(double *Z, const double StildeSq, const double D, const double tauTilde, double gamma)
{
  // Rootfind data
  double x0(*Z);
  double x1(x0 + EPS);
  double x2;
  double f0(residualParallel(x0, StildeSq, D, tauTilde, gamma));
  double f1(residualParallel(x1, StildeSq, D, tauTilde, gamma));
  int iter;

  for (iter=0; iter<MAXITER; iter++) {
    if (fabs(f0) < TOL) {
      *Z = x0;
      return 1;
    }

    x2 = x1 - f1 * (x1 - x0) / (f1 - f0);
    x1 = x0;
    x0 = x2;
    f1 = f0;
    f0 = residualParallel(x0, StildeSq, D, tauTilde, gamma);
  }

  return 0;
}

__device__
static double residualParallel(const double Z, const double S_sqrd, const double D, const double tau, double gamma)
{
  // Decalre variables
  double v_sqrd, E, W, rho, p, n, rho_plus_p, resid;

  E = tau + D;
  v_sqrd = S_sqrd / ((E + Z)*(E + Z));

  // Sanity check
  if (v_sqrd >= 1.0 || Z < 0) return 1.0e6;

  W = 1/sqrt(1 - v_sqrd);
  n = D / W;
  rho_plus_p = (E + Z)/(W*W);
  p = (rho_plus_p - n)*((gamma-1)/gamma);
  rho = rho_plus_p - p;

  // Second sanity check
  if (rho < 0 || p < 0 || W < 1 ) return 1.0e6;

  // Values are physical, compute residual
  resid = p - Z;
  return resid;

}


__device__
void ISCE_D::getPrimitiveVarsSingleCell(double * cons, double * prims, double * aux)
{
    // Sbarsq, tauBar
    aux[Aux::S_sqrd] = cons[Cons::S1] * cons[Cons::S1] + cons[Cons::S2] * cons[Cons::S2] + cons[Cons::S3] * cons[Cons::S3];
 
    // Solve
    newtonParallel(&prims[Prims::p], aux[Aux::S_sqrd], cons[Cons::D], cons[Cons::Tau], args->gamma);

    double E = cons[Cons::Tau] + cons[Cons::D];
    
    // vsq
    aux[Aux::vsqrd] = aux[Aux::S_sqrd] / ((E + prims[Prims::p]) * (E + prims[Prims::p]));
    // W
    aux[Aux::W] = 1.0 / sqrt(1 - aux[Aux::vsqrd]);
    // rho
    prims[Prims::n] = cons[Cons::D] / aux[Aux::W];
    // rho_plus_p
    double rho_plus_p = (E + prims[Prims::p]) / (aux[Aux::W] * aux[Aux::W]);
    // p  
    prims[Prims::p] = (rho_plus_p - prims[Prims::n]) / ((args->gamma-1)/args->gamma);
    // rho
    prims[Prims::rho] = rho_plus_p - prims[Prims::p];    
    // vx, vy, vz
    prims[Prims::v1] = cons[Cons::S1] / (rho_plus_p*aux[Aux::W] * aux[Aux::W]);
    prims[Prims::v1] = cons[Cons::S2] / (rho_plus_p*aux[Aux::W] * aux[Aux::W]);
    prims[Prims::v1] = cons[Cons::S3] / (rho_plus_p*aux[Aux::W] * aux[Aux::W]);

    aux[Aux::e] = prims[Prims::p] / (prims[Prims::n]*(args->gamma-1));
    aux[Aux::T] = prims[Prims::p] / prims[Prims::n];
    aux[Aux::h] = 1 + aux[Aux::e] + prims[Prims::p] / prims[Prims::n];
}

__device__
void ISCE_D::sourceTermSingleCell(double *cons, double *prims, double *aux, double *source)
{
  // D
  source[0] = 0.0;
  // S1,2,3
  source[1] = 0.0; 
  source[2] = 0.0;
  source[3] = 0.0; 
  // Tau
  source[4] = 0.0;
}
