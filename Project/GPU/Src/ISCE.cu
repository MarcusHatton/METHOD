#include "hip/hip_runtime.h"
//! Special relativistic dissipative hydro model - ISCE
/*!
    This script contains the function definitions for the ISCE model. The form
  of the quations has been taken from Dionysopoulou and we use a divergence cleaning method
  taken from Muddle.
    For detailed documentation about the methods contained herein, see ISCE.h
  and model.h.
*/

#include <stdexcept>
#include <cstdio>
#include "ISCE.h"
#include "cminpack.h"
#include "cudaErrorCheck.h"

#define TOL 1.0e-12
#define EPS 1.0e-4
#define MAXITER 50

// Macro for getting array index
#define ID(variable, idx, jdx, kdx) ((variable)*(d->Nx)*(d->Ny)*(d->Nz) + (idx)*(d->Ny)*(d->Nz) + (jdx)*(d->Nz) + (kdx))
#define IDCons(var, idx, jdx, kdx) ( (var) + (idx)*(d->Ncons)*(d->Nz)*(d->Ny) + (jdx)*(d->Ncons)*(d->Nz) + (kdx)*(d->Ncons)  )
#define IDPrims(var, idx, jdx, kdx) ( (var) + (idx)*(d->Nprims)*(d->Nz)*(d->Ny) + (jdx)*(d->Nprims)*(d->Nz) + (kdx)*(d->Nprims)  )
#define IDAux(var, idx, jdx, kdx) ( (var) + (idx)*(d->Naux)*(d->Nz)*(d->Ny) + (jdx)*(d->Naux)*(d->Nz) + (kdx)*(d->Naux)  )

// C2P residual and rootfinder (Serial)
static double residual(const double, const double, const double, const double, double);
static int newton(double *Z, const double StildeSq, const double D, const double tauTilde, double gamma, int i, int j, int k);

// C2P residual and rootfinder (Parallel)
__device__
static double residualParallel(const double Z, const double StildeSq, const double D, const double tauTilde, double gamma);
__device__
static int newtonParallel(double *Z, const double StildeSq, const double D, const double tauTilde, double gamma);
__global__
static void getPrimitiveVarsParallel(double *cons, double *prims, double *aux, double *guess, int stream, double gamma, double sigma, int Ncons, int Nprims, int Naux, int origWidth, int streamWidth);

ISCE::ISCE() : Model()
{
  modType_t = ModelType::ISCE;
  this->Ncons = 5;
  this->Nprims = 16;
  this->Naux = 59;

  hipHostAlloc((void **)&singleCons, sizeof(double) * this->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singlePrims, sizeof(double) * this->Nprims,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singleAux, sizeof(double) * this->Naux,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singleSource, sizeof(double) * this->Ncons,
                hipHostMallocPortable);
}

ISCE::~ISCE()
{
  // Free up
  hipHostFree(singleCons);
  hipHostFree(singlePrims);
  hipHostFree(singleAux);
  hipHostFree(singleSource);

  delete c2pArgs;
}

ISCE::ISCE(Data * data) : Model(data)
{
  modType_t = ModelType::ISCE;
  this->Ncons = (this->data)->Ncons = 5;
  this->Nprims = (this->data)->Nprims = 16;
  this->Naux = (this->data)->Naux = 59;

  // 0
  this->data->primsLabels.push_back("v1");   this->data->primsLabels.push_back("v2");
  this->data->primsLabels.push_back("v3");
  // 3
  this->data->primsLabels.push_back("p");   this->data->primsLabels.push_back("rho");
  this->data->primsLabels.push_back("n");   
  // 6
  this->data->primsLabels.push_back("q1");  this->data->primsLabels.push_back("q2");
  this->data->primsLabels.push_back("q3");
  // 9
  this->data->primsLabels.push_back("Pi");  
  // 10
  this->data->primsLabels.push_back("pi11");   this->data->primsLabels.push_back("pi12");
  this->data->primsLabels.push_back("pi13");  this->data->primsLabels.push_back("pi22");
  this->data->primsLabels.push_back("pi23");  this->data->primsLabels.push_back("pi33");

  // 0
  this->data->auxLabels.push_back("h");     this->data->auxLabels.push_back("T");
  this->data->auxLabels.push_back("e");     this->data->auxLabels.push_back("W");
  // 4
  this->data->auxLabels.push_back("q0");    this->data->auxLabels.push_back("qv");
  this->data->auxLabels.push_back("pi00");  this->data->auxLabels.push_back("pi01");
  this->data->auxLabels.push_back("pi02");  this->data->auxLabels.push_back("pi03");
  this->data->auxLabels.push_back("Theta"); this->data->auxLabels.push_back("vsqrd");
  // 12
  this->data->auxLabels.push_back("q1NS");  this->data->auxLabels.push_back("q2NS");
  this->data->auxLabels.push_back("q3NS");
  // 15
  this->data->auxLabels.push_back("PiNS");    
  // 16
  this->data->auxLabels.push_back("pi11NS"); this->data->auxLabels.push_back("pi12NS");
  this->data->auxLabels.push_back("pi13NS"); this->data->auxLabels.push_back("pi22NS");
  this->data->auxLabels.push_back("pi23NS"); this->data->auxLabels.push_back("pi33NS");
  // 22
  this->data->auxLabels.push_back("q1LO");  this->data->auxLabels.push_back("q2LO");
  this->data->auxLabels.push_back("qLO");
  // 25
  this->data->auxLabels.push_back("PiLO");    
  // 26
  this->data->auxLabels.push_back("pi11LO"); this->data->auxLabels.push_back("pi12LO");
  this->data->auxLabels.push_back("pi13LO"); this->data->auxLabels.push_back("pi22LO");
  this->data->auxLabels.push_back("pi23LO"); this->data->auxLabels.push_back("pi33LO");
  // 32
  this->data->auxLabels.push_back("a1");     this->data->auxLabels.push_back("a2");   
  this->data->auxLabels.push_back("a3");

  // 35
  this->data->auxLabels.push_back("dtp");  this->data->auxLabels.push_back("dtrho");
  this->data->auxLabels.push_back("dtn");
  // 38
  this->data->auxLabels.push_back("dtv1");
  this->data->auxLabels.push_back("dtv2");  this->data->auxLabels.push_back("dtv3");
  // 41
  this->data->auxLabels.push_back("dtW");   this->data->auxLabels.push_back("dtT"); 
  // 43
  this->data->auxLabels.push_back("dtq1NS");  this->data->auxLabels.push_back("dtq2NS");
  this->data->auxLabels.push_back("dtq3NS");
  // 46
  this->data->auxLabels.push_back("dtPiNS");    
  // 47
  this->data->auxLabels.push_back("dtpi11NS"); this->data->auxLabels.push_back("dtpi12NS");
  this->data->auxLabels.push_back("dtpi13NS"); this->data->auxLabels.push_back("dtpi22NS");
  this->data->auxLabels.push_back("dtpi23NS"); this->data->auxLabels.push_back("dtpi33NS");
  // 53
  this->data->auxLabels.push_back("dtD"); this->data->auxLabels.push_back("dtS1");
  this->data->auxLabels.push_back("dtS2"); this->data->auxLabels.push_back("dtS3");
  this->data->auxLabels.push_back("dtTau");  this->data->auxLabels.push_back("dtE"); 

  // Single cell work arrays
  hipHostAlloc((void **)&singleCons, sizeof(double) * this->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singlePrims, sizeof(double) * this->Nprims,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singleAux, sizeof(double) * this->Naux,
                hipHostMallocPortable);
  hipHostAlloc((void **)&singleSource, sizeof(double) * this->Ncons,
                hipHostMallocPortable);

  c2pArgs = new C2PArgs(this->data);
}

void ISCE::fluxVector(double *cons, double *prims, double *aux, double *f, const int dir)
{
  // Syntax
  Data * d(this->data);

  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        // Dv
        f[ID(0, i, j, k)] = cons[ID(Cons::D, i, j, k)]*prims[ID(dir, i, j, k)];
        // Sv + ..
        for (int nvar(0); nvar < 3; nvar++) {
          f[ID(1+nvar, i, j, k)] = cons[ID(Cons::S1+nvar, i, j, k)]*prims[ID(dir, i, j, k)]; // + ( prims[ID(Prims::q1+dir, i, j, k)] * prims[ID(Prims::v1+nvar, i, j, k)]  
            // - aux[ID(Aux::qv, i, j, k)]*prims[ID(Prims::v1+nvar, i, j, k)]*prims[ID(Prims::v1+dir, i, j, k)] ) * aux[ID(Aux::W, i, j, k)];
          // (p+Pi)delta_ij
          if (dir == nvar) {
            f[ID(1+nvar, i, j, k)] += (prims[ID(Prims::p, i, j, k)]); // + prims[ID(Prims::Pi, i, j, k)]);
          }
        }
        /*
        //  pi^i_j  
        if (dir == 0) {
          for (int nvar(0); nvar < 3; nvar++) {
            f[ID(1+nvar, i, j, k)] += prims[ID(Prims::pi11+nvar, i, j, k)];
          }
        } else if (dir == 1) {
          f[ID(1, i, j, k)] += prims[ID(Prims::pi12, i, j, k)];
          f[ID(2, i, j, k)] += prims[ID(Prims::pi22, i, j, k)];
          f[ID(3, i, j, k)] += prims[ID(Prims::pi23, i, j, k)];
        } else if (dir == 2) {
          f[ID(1, i, j, k)] += prims[ID(Prims::pi13, i, j, k)];
          f[ID(2, i, j, k)] += prims[ID(Prims::pi23, i, j, k)];
          f[ID(3, i, j, k)] += prims[ID(Prims::pi33, i, j, k)];
        } else {
          throw std::runtime_error("Flux direction is not 0, 1 or 2");
        }
        */

        // (Tau+p)*v + ...
        f[ID(4, i, j, k)] = (cons[ID(Cons::Tau, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * prims[ID(dir, i, j, k)];
          // + (prims[ID(Prims::q1+dir, i, j, k)] - aux[ID(Aux::qv, i, j, k)]*prims[ID(Prims::v1+dir, i, j, k)])*aux[ID(Aux::W, i, j, k)]
          // + aux[ID(Aux::pi01+dir, i, j, k)];
      } // End k loop
    } // End j loop
  } // End i loop
}

void ISCE::sourceTermSingleCell(double *cons, double *prims, double *aux, double *source, int i, int j, int k)
{
  // Syntax
  Data * d(this->data);

  // D
  source[0] = 0.0;
  // S1,2,3
  source[1] = 0.0; 
  source[2] = 0.0;
  source[3] = 0.0; 
  // Tau
  source[4] = 0.0;

}

void ISCE::sourceTerm(double *cons, double *prims, double *aux, double *source)
{
  for (int i(0); i < this->data->Nx; i++) {
    for (int j(0); j < this->data->Ny; j++) {
      for (int k(0); k < this->data->Nz; k++) {

        // D
        source[ID(D, i, j, k)] = 0.0;
        // S1,2,3
        source[ID(S1, i, j, k)] = 0.0;
        source[ID(S2, i, j, k)] = 0.0;
        source[ID(S3, i, j, k)] = 0.0;
        // Tau
        source[ID(Tau, i, j, k)] = 0.0;        
      }
    }
  }
}

void ISCE::getPrimitiveVarsSingleCell(double *cons, double *prims, double *aux, int i, int j, int k)
{
  // Syntax
  Data * d(this->data);

  // Hybrd1 set-up
  Args args;                      // Additional arguments structure
  const int sys_size(1);                     // Size of system
  double sol[sys_size];                      // Guess and solution vector
  double res[sys_size];                      // Residual/fvec vector
  int info;                           // Rootfinder flag
  const double tol = 1e-5;          // Tolerance of rootfinder
  const int lwa = 8;                 // Length of work array = n * (3*n + 13) / 2
  double wa[lwa];                     // Work array

  // Set additional args for rootfind
  args.D_rf = cons[Cons::D];
  args.S1_rf = cons[Cons::S1];
  args.S2_rf = cons[Cons::S2];
  args.S3_rf = cons[Cons::S3];
  args.Tau_rf = cons[Cons::Tau];
  args.gamma = d->gamma;
  
  sol[0] = prims[Prims::p]; // Guess the pressure

  // Solve residual = 0
  info = __cminpack_func__(hybrd1) (&ISCEresidual, &args, sys_size, sol, res,
                                    tol, wa, lwa);
  // If root find fails, add failed cell to the list
  if (info!=1) {
    //printf("C2P single cell failed for cell (%d, %d, %d), hybrd returns info=%d\n", i, j, k, info);
    throw std::runtime_error("C2P could not converge.\n");
  }
  aux[Aux::vsqrd] = (cons[Cons::S1]*cons[Cons::S1] + cons[Cons::S2]*cons[Cons::S2] 
                      + cons[Cons::S3]*cons[Cons::S3] - sol[3])
                      /((cons[Cons::Tau] + cons[Cons::D] + sol[0])*(cons[Cons::Tau]  + cons[Cons::D] + sol[0]));
  aux[Aux::W] = 1 / sqrt((1-aux[Aux::vsqrd]));
  prims[Prims::n] = cons[Cons::D] / aux[Aux::W];
  double rho_plus_p = (cons[Cons::Tau] + cons[Cons::D] + sol[0])/(aux[Aux::W]*aux[Aux::W]);
  prims[Prims::v1] = cons[Cons::S1]/(rho_plus_p*aux[Aux::W]*aux[Aux::W]);
  prims[Prims::v2] = cons[Cons::S2]/(rho_plus_p*aux[Aux::W]*aux[Aux::W]);
  prims[Prims::v3] = cons[Cons::S3]/(rho_plus_p*aux[Aux::W]*aux[Aux::W]);
  prims[Prims::p] = (rho_plus_p - prims[Prims::n])*((d->gamma-1)/d->gamma);
  prims[Prims::rho] = rho_plus_p - prims[Prims::p];
  
  aux[Aux::e] = prims[Prims::p] / (prims[Prims::n]*(d->gamma-1));
  aux[Aux::T] = prims[Prims::p] / prims[Prims::n];     
  aux[Aux::h] = 1 + aux[Aux::e] + prims[Prims::p] / prims[Prims::n];

}

void ISCE::getPrimitiveVars(double *cons, double *prims, double *aux)
{
  // Syntax
  Data * d(this->data);

  // First need to copy data to the device
  // A single cell requires all cons variables and aux10 to start the guessing
  // Rearrange data into host arrays ready for copying
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        for (int var(0); var < d->Ncons; var++) {
          c2pArgs->cons_h[IDCons(var, i, j, k)] = cons[ID(var, i, j, k)];
        }
        c2pArgs->guess_h[ID(0, i, j, k)] = aux[ID(10, i, j, k)];
      }
    }
  }

  // Data is in correct order, now stream data to the device
  for (int i(0); i < c2pArgs->Nstreams; i++) {
    // Which cell is at the left bound?
    int lcell(i * c2pArgs->streamWidth);
    // Which cell is at the right bound?
    int rcell(lcell + c2pArgs->streamWidth);
    if (rcell > d->Ncells) rcell = d->Ncells;
    // Memory size to copy in
    int width(rcell - lcell);
    int inMemsize(width * sizeof(double));

    // Send stream's data
    gpuErrchk( hipMemcpyAsync(c2pArgs->cons_d[i], c2pArgs->cons_h + lcell*d->Ncons, inMemsize*d->Ncons, hipMemcpyHostToDevice, c2pArgs->stream[i]) );
    gpuErrchk( hipMemcpyAsync(c2pArgs->guess_d[i], c2pArgs->guess_h + lcell, inMemsize, hipMemcpyHostToDevice, c2pArgs->stream[i]) );

    // Call kernel and operate on data
    getPrimitiveVarsParallel <<< c2pArgs->bpg, c2pArgs->tpb,
        c2pArgs->tpb * c2pArgs->cellMem, c2pArgs->stream[i] >>> (c2pArgs->cons_d[i],
        c2pArgs->prims_d[i], c2pArgs->aux_d[i], c2pArgs->guess_d[i], i, d->gamma, d->sigma, d->Ncons,
        d->Nprims, d->Naux, c2pArgs->streamWidth, width);


    // Copy all data back
    gpuErrchk( hipMemcpyAsync(c2pArgs->prims_h + lcell*d->Nprims, c2pArgs->prims_d[i], inMemsize*d->Nprims, hipMemcpyDeviceToHost, c2pArgs->stream[i]) );
    gpuErrchk( hipMemcpyAsync(c2pArgs->aux_h + lcell*d->Naux, c2pArgs->aux_d[i], inMemsize*d->Naux, hipMemcpyDeviceToHost, c2pArgs->stream[i]) );
  }
  gpuErrchk( hipDeviceSynchronize() );

  // Rearrange data back into arrays
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {

        for (int var(0); var < d->Nprims; var++) {
          prims[ID(var, i, j, k)] = c2pArgs->prims_h[IDPrims(var, i, j, k)];
        }
        for (int var(0); var < d->Naux; var++) {
          aux[ID(var, i, j, k)] = c2pArgs->aux_h[IDAux(var, i, j, k)];
        }
      }
    }
  }
}

void ISCE::primsToAll(double *cons, double *prims, double *aux)
{
  // Syntax
  Data * d(this->data);

  // printf("Calling primsToAll\n");

  // W, q_kv^k, pi^0_0
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        aux[ID(Aux::vsqrd, i, j, k)] = prims[ID(Prims::v1, i, j, k)]*prims[ID(Prims::v1, i, j, k)] 
                                  + prims[ID(Prims::v2, i, j, k)]*prims[ID(Prims::v2, i, j, k)] 
                                  + prims[ID(Prims::v3, i, j, k)]*prims[ID(Prims::v3, i, j, k)];
        aux[ID(Aux::W, i, j, k)] = 1 / sqrt( 1 - aux[ID(Aux::vsqrd, i, j, k)] );
        aux[ID(Aux::e, i, j, k)] = prims[ID(Prims::p, i, j, k)] / (prims[ID(Prims::n, i, j, k)]*(d->gamma-1));
        prims[ID(Prims::rho, i, j, k)] = prims[ID(Prims::n, i, j, k)]*(1+aux[ID(Aux::e, i, j, k)]);
        aux[ID(Aux::T, i, j, k)] = prims[ID(Prims::p, i, j, k)] / prims[ID(Prims::n, i, j, k)];
        aux[ID(Aux::h, i, j, k)] = 1 + aux[ID(Aux::e, i, j, k)] + prims[ID(Prims::p, i, j, k)] / prims[ID(Prims::n, i, j, k)];
      }
    }
  }

  // Conserveds are now EulerSR form (no dissipation)
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        // D
        cons[ID(Cons::D, i, j, k)] = prims[ID(Prims::n, i, j, k)] * aux[ID(Aux::W, i, j, k)];
        // S1,2,3
        cons[ID(Cons::S1, i, j, k)] = (prims[ID(Prims::rho, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * aux[ID(Aux::W, i, j, k)]*aux[ID(Aux::W, i, j, k)] * prims[ID(Prims::v1, i, j, k)]; 
        cons[ID(Cons::S2, i, j, k)] = (prims[ID(Prims::rho, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * aux[ID(Aux::W, i, j, k)]*aux[ID(Aux::W, i, j, k)] * prims[ID(Prims::v2, i, j, k)]; 
        cons[ID(Cons::S3, i, j, k)] = (prims[ID(Prims::rho, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * aux[ID(Aux::W, i, j, k)]*aux[ID(Aux::W, i, j, k)] * prims[ID(Prims::v3, i, j, k)];
        // Tau
        cons[ID(Cons::Tau, i, j, k)] = (prims[ID(Prims::rho, i, j, k)] + prims[ID(Prims::p, i, j, k)]) * aux[ID(Aux::W, i, j, k)]*aux[ID(Aux::W, i, j, k)] 
        - (prims[ID(Prims::p, i, j, k)] + prims[ID(Prims::n, i, j, k)] * aux[ID(Aux::W, i, j, k)]);
      }  
    }
  }
}

static double residual(const double Z, const double StildeSq, const double D, const double tauTilde, double gamma)
{
  // Decalre variables
  double vsq, W, rho, h, p, resid;

  vsq = StildeSq / (Z * Z);

  // Sanity check
  if (vsq >= 1.0 || Z < 0) return 1.0e6;

  // Continue
  W = 1 / sqrt(1 - vsq);
  rho = D / W;
  h = Z / (rho * W * W);
  p = (gamma - 1) * (h - rho) / gamma;

  // Second sanity check
  if (rho < 0 || p < 0 || W < 1 || h < 0) return 1.0e6;

  // Values are physical, compute residual
  resid = (1 - (gamma - 1) / (W * W * gamma)) * Z + ((gamma - 1) /
          (W * gamma) - 1) * D - tauTilde;

  return resid;

}


static int newton(double *Z, const double StildeSq, const double D, const double tauTilde, double gamma, int i, int j, int k)
{
  // Rootfind data
  double bestX;
  double x0(*Z);
  double eps(EPS);
  double x1(x0 + eps);
  double tol(TOL);
  double x2;
  double bestF;
  double f0(residual(x0, StildeSq, D, tauTilde, gamma));
  double f1(residual(x1, StildeSq, D, tauTilde, gamma));
  int iter;
  int maxiter(MAXITER);
  int found(0);

  // If root can not be found return the best so far
  bestX = x0; bestF = f0;
  for (iter=0; iter<maxiter; iter++) {
    if (fabs(f0) < tol) {
      *Z = x0;
      found = 1;
      break;
    }

    x2 = x1 - f1 * (x1 - x0) / (f1 - f0);
    x1 = x0;
    x0 = x2;
    f1 = f0;
    f0 = residual(x0, StildeSq, D, tauTilde, gamma);
    if (fabs(f0) < fabs(bestF)) {
      bestX = x0;
      bestF = f0;
    }
  }

  if (!found) {
    // Store result of Z=rho*h*W**2
    *Z = bestX;
    char s[200];
    sprintf(s, "C2P could not converge in cell (%d, %d, %d)\n", i, j, k);

    throw std::runtime_error(s);
  }
  return 1;
}

// /*!
//     This is the device version of the getPrimitiveVars that takes a streams data
//     and computes the rest of the prims and aux vars. This is called when
//     ISCE::getPrimitiveVars is required, i.e. all cells need to be found.
// */
__global__
static void getPrimitiveVarsParallel(double *streamCons, double *streamPrims, double *streamAux, double *guess, int stream, double gamma, double sigma, int Ncons, int Nprims, int Naux, int origWidth, int streamWidth)
{
  // First need thread indicies
  const int tID(threadIdx.x);                     //!< thread index (in block)
  const int lID(tID + blockIdx.x * blockDim.x);   //!< local index (in stream)
  // const int gID(lID + stream * origWidth);        //!< global index (in domain)
  // Allocate shared memory
  extern __shared__ double sharedArray [];
  double * cons = &sharedArray[tID * (Ncons + Nprims + Naux)];
  double * prims = &cons[Ncons];
  double * aux = &prims[Nprims];

  if (lID < streamWidth) {

    // Load conserved vector into shared memory, and the initial guess
    for (int i(0); i < Ncons; i++) cons[i] = streamCons[lID * Ncons + i];
    aux[10] = guess[lID];

    // Set Bx/y/z and Ex/y/z field in prims
    prims[5] = cons[5]; prims[6] = cons[6]; prims[7] = cons[7];
    prims[8] = cons[8]; prims[9] = cons[9]; prims[10] = cons[10];

    // Bsq, Esq
    aux[7] = cons[5] * cons[5] + cons[6] * cons[6] + cons[7] * cons[7];
    aux[8] = cons[8] * cons[8] + cons[9] * cons[9] + cons[10] * cons[10];

    // Sbarx, Sbary, Sbarz
    aux[12] = cons[1] - (cons[9] * cons[7] - cons[10] * cons[6]);
    aux[13] = cons[2] - (cons[10] * cons[5] - cons[8] * cons[7]);
    aux[14] = cons[3] - (cons[8] * cons[6] - cons[9] * cons[5]);
    // Sbarsq, tauBar
    aux[15] = aux[12] * aux[12] + aux[13] * aux[13] + aux[14] * aux[14];
    aux[16] = cons[4] - 0.5 * (aux[7] + aux[8]);

    // Solve
    newtonParallel(&aux[10], aux[15], cons[0], aux[16], gamma);

    // vsq
    aux[9] = aux[15] / (aux[10] * aux[10]);

    // W
    aux[1] = 1.0 / sqrt(1 - aux[9]);
    // rho
    prims[0] = cons[0] / aux[1];
    // h
    aux[0] = aux[10] / (prims[0] * aux[1] * aux[1]);
    // e
    aux[2] = (aux[0] - 1) / gamma;
    // c
    aux[3] = sqrt((aux[2] * gamma * (gamma - 1)) / aux[0]);
    // p
    prims[4] = prims[0] * aux[2] * (gamma - 1);
    // vx, vy, vz
    prims[1] = aux[12] / aux[10];
    prims[2] = aux[13] / aux[10];
    prims[3] = aux[14] / aux[10];
    // vE
    aux[11] = prims[1] * cons[8] + prims[2] * cons[9] + prims[3] * cons[10];
    // Jx, Jy, Jz
    aux[4] = cons[13] * prims[1] + aux[1] * sigma * (cons[8] + (prims[2] * cons[7] -
             prims[3] * cons[6]) - aux[11] * prims[1]);
    aux[5] = cons[13] * prims[2] + aux[1] * sigma * (cons[9] + (prims[3] * cons[5] -
             prims[1] * cons[7]) - aux[11] * prims[2]);
    aux[6] = cons[13] * prims[3] + aux[1] * sigma * (cons[10] + (prims[1] * cons[6] -
             prims[2] * cons[5]) - aux[11] * prims[3]);

  }

  // Copy data back from shared memory into device arrays
  for (int i(0); i < Nprims; i++) streamPrims[lID * Nprims + i] = prims[i];
  for (int i(0); i < Naux; i++) streamAux[lID * Naux + i] = aux[i];

}

__device__
static int newtonParallel(double *Z, const double StildeSq, const double D, const double tauTilde, double gamma)
{
  // Rootfind data
  double x0(*Z);
  double x1(x0 + EPS);
  double x2;
  double f0(residualParallel(x0, StildeSq, D, tauTilde, gamma));
  double f1(residualParallel(x1, StildeSq, D, tauTilde, gamma));
  int iter;

  for (iter=0; iter<MAXITER; iter++) {
    if (fabs(f0) < TOL) {
      *Z = x0;
      return 1;
    }

    x2 = x1 - f1 * (x1 - x0) / (f1 - f0);
    x1 = x0;
    x0 = x2;
    f1 = f0;
    f0 = residualParallel(x0, StildeSq, D, tauTilde, gamma);
  }

  return 0;
}

__device__
static double residualParallel(const double Z, const double StildeSq, const double D, const double tauTilde, double gamma)
{
  // Declare variables
  double  W;

  // Sanity check
  if (Z < 0) return 1.0e6;

  // Continue
  W = 1 / sqrt(1 - (StildeSq / (Z * Z)));

  // Values are physical, compute residual
  return (1 - (gamma - 1) / (W * W * gamma)) * Z + ((gamma - 1) /
          (W * gamma) - 1) * D - tauTilde;

}


__device__
void ISCE_D::getPrimitiveVarsSingleCell(double * cons, double * prims, double * aux)
{

    // Set Bx/y/z and Ex/y/z field in prims
    prims[5] = cons[5]; prims[6] = cons[6]; prims[7] = cons[7];
    prims[8] = cons[8]; prims[9] = cons[9]; prims[10] = cons[10];

    // Bsq, Esq
    aux[7] = cons[5] * cons[5] + cons[6] * cons[6] + cons[7] * cons[7];
    aux[8] = cons[8] * cons[8] + cons[9] * cons[9] + cons[10] * cons[10];

    // Sbarx, Sbary, Sbarz
    aux[12] = cons[1] - (cons[9] * cons[7] - cons[10] * cons[6]);
    aux[13] = cons[2] - (cons[10] * cons[5] - cons[8] * cons[7]);
    aux[14] = cons[3] - (cons[8] * cons[6] - cons[9] * cons[5]);
    // Sbarsq, tauBar
    aux[15] = aux[12] * aux[12] + aux[13] * aux[13] + aux[14] * aux[14];
    aux[16] = cons[4] - 0.5 * (aux[7] + aux[8]);


    // Solve
    newtonParallel(&aux[10], aux[15], cons[0], aux[16], args->gamma);

    // vsq
    aux[9] = aux[15] / (aux[10] * aux[10]);
    // W
    aux[1] = 1.0 / sqrt(1 - aux[9]);
    // rho
    prims[0] = cons[0] / aux[1];
    // h
    aux[0] = aux[10] / (prims[0] * aux[1] * aux[1]);
    // e
    aux[2] = (aux[0] - 1) / args->gamma;
    // c
    aux[3] = sqrt((aux[2] * args->gamma * (args->gamma - 1)) / aux[0]);
    // p
    prims[4] = prims[0] * aux[2] * (args->gamma - 1);
    // vx, vy, vz
    prims[1] = aux[12] / aux[10];
    prims[2] = aux[13] / aux[10];
    prims[3] = aux[14] / aux[10];
    // vE
    aux[11] = prims[1] * cons[8] + prims[2] * cons[9] + prims[3] * cons[10];
    // Jx, Jy, Jz
    aux[4] = cons[13] * prims[1] + aux[1] * args->sigma * (cons[8] + (prims[2] * cons[7] -
             prims[3] * cons[6]) - aux[11] * prims[1]);
    aux[5] = cons[13] * prims[2] + aux[1] * args->sigma * (cons[9] + (prims[3] * cons[5] -
             prims[1] * cons[7]) - aux[11] * prims[2]);
    aux[6] = cons[13] * prims[3] + aux[1] * args->sigma * (cons[10] + (prims[1] * cons[6] -
             prims[2] * cons[5]) - aux[11] * prims[3]);
}

__device__
void ISCE_D::sourceTermSingleCell(double *cons, double *prims, double *aux, double *source)
{
  // D
  source[0] = 0.0;
  // S1,2,3
  source[1] = 0.0; 
  source[2] = 0.0;
  source[3] = 0.0; 
  // Tau
  source[4] = 0.0;
}
